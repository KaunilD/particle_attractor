#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ float dot(float4 a, float4 b){
	return a.x*b.x + a.y*b.y + a.z*b.z + a.w*b.w;
}

__global__ void kernel_update(
	float4* d_positions, float4* d_velocities,
	float4* h_positions, float4* h_velocities,
	float* d_masses, size_t numel) {

	size_t col = threadIdx.x + blockIdx.x * blockDim.x;
	if (col >= numel) { return; }

	float4 distance = d_positions[col];
	float magnitude = sqrt(
		distance.x * distance.x +
		distance.y * distance.y +
		distance.z * distance.z
	);
	magnitude			= min(30.0f, max(magnitude, 5.0f));
	float force			= (0.4 * d_masses[0]) / (magnitude);
	float coeff			= rsqrt(dot(distance, distance))*force;

	float4 acceleration	= make_float4(
		coeff*-distance.x, 
		coeff*-distance.y, 
		coeff*-distance.z, 
		coeff*-distance.w
	);	
	
	
	d_velocities[col]	= make_float4(
		d_velocities[col].x +acceleration.x * 0.05f,
		d_velocities[col].y +acceleration.y * 0.05f,
		d_velocities[col].z +acceleration.z * 0.05f,
		d_velocities[col].w +acceleration.w * 0.05f
	);

	d_positions[col]	= make_float4(
		d_positions[col].x + d_velocities[col].x * 0.05f,
		d_positions[col].y + d_velocities[col].y * 0.05f,
		d_positions[col].z + d_velocities[col].z * 0.05f,
		d_positions[col].w + d_velocities[col].w * 0.05f
	);

}

void launch_kernel_update(
	float4* d_positions, float4* d_velocities, 
	float4* h_positions, float4* h_velocities, 
	float* masses, int numel) {
	dim3 blockSize(32, 1, 1);
	dim3 gridSize(1);
	gridSize.x = numel / blockSize.x;
	kernel_update << <gridSize, blockSize >> > (
		d_positions, d_velocities,
		h_positions, h_velocities,
		masses, numel
	);

	hipDeviceSynchronize();
}