#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cuda/utils.hpp"

#define KERN_RADIUS 5

__constant__ float gaussian_kernel[KERN_RADIUS * KERN_RADIUS];
__constant__ float hs_kernel[9];

__device__ size_t GIDX(size_t row, size_t col, int H, int W) {
	return row * W + col;
}


void setGaussian(float const* elements, const int count) {
	if (count != 25) {
		std::cerr << "Only 5x5 gaussian kernel supported!" << std::endl;
		return;
	}
	CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(gaussian_kernel), elements, count * sizeof(float)));
}

void setHS(float const* elements, const int count) {
	if (count != 9) {
		std::cerr << "Only 3x3 horn-schunke kernel supported!" << std::endl;
		return;
	}
	CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(hs_kernel), elements, count * sizeof(float)));
}

__global__ void kernel_grey(
	float4* d_Iin,
	float* d_Iout,
	int numel
) {

	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	if (col >= numel) {
		return;
	}

	float4 pixel = d_Iin[col];

	d_Iout[col] = 0.2989f * (pixel.x) + 0.5870f * (pixel.y) + 0.1140f * (pixel.z);
}

void launch_gray(
	float4* d_Iin,
	float* d_Iout,
	int numel
) {


	dim3 blockSize = { 320 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(numel / blockSize.x))
	};

	kernel_grey << <gridSize, blockSize >> > (
		d_Iin, d_Iout, numel
		);

	CUDACHECK(hipDeviceSynchronize());
}


__global__ void kernel_partials(
	float* d_f1ptr,
	float* d_f1dx, float* d_f1dy,
	int H, int W
) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;

	size_t idx = GIDX(row, col, H, W);
	if (row >= H || row <= 1 || col >= W || col <= 1) {
		return;
	}

	float gray_x1 = d_f1ptr[GIDX(row, col - 1, H, W)];
	float gray_x2 = d_f1ptr[GIDX(row, col + 1, H, W)];

	float gray_y1 = d_f1ptr[GIDX(row - 1, col, H, W)];
	float gray_y2 = d_f1ptr[GIDX(row + 1, col, H, W)];

	d_f1dx[idx] = (gray_x2 - gray_x1) / 2.0f;
	d_f1dy[idx] = (gray_y2 - gray_y1) / 2.0f;
}


void launch_partials(
	float* d_f1ptr,
	float* d_f1dx, float* d_f1dy,
	int H, int W
) {


	dim3 blockSize = { 32, 32 };
	dim3 gridSize = {
	static_cast<unsigned int>(ceil(W / blockSize.x)),
	static_cast<unsigned int>(ceil(H / blockSize.y))
	};

	kernel_partials << <gridSize, blockSize >> > (
		d_f1ptr, d_f1dx, d_f1dy, H, W
		);

	CUDACHECK(hipDeviceSynchronize());
}


__global__ void kernel_sub(float* d_f1ptr, float* d_f2ptr, float* d_dt, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = GIDX(row, col, H, W);

	if (row >= H || col >= W) {
		return;
	}

	d_dt[idx] = d_f2ptr[idx] - d_f1ptr[idx];

}


void launch_sub(float* d_f1ptr, float* d_f2ptr, float* d_dt, int H, int W) {

	dim3 blockSize = { 32, 32 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(W / blockSize.x)),
		static_cast<unsigned int>(ceil(H / blockSize.y))
	};

	kernel_sub << <gridSize, blockSize >> > (
		d_f1ptr, d_f2ptr, d_dt, H, W
		);

	CUDACHECK(hipDeviceSynchronize());
}

__global__ void kernel_optflow(float* d_dx1, float* d_dy1, float* d_dx2, float* d_dy2, float* d_dt, float4* uv, float4* uv1, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = GIDX(row, col, H, W);

	if (row >= H || row <= 1 || col >= W || col <= 1) {
		return;
	}

	float grad_x = (d_dx1[idx] + d_dx2[idx]) / 2.0f;
	float grad_y = (d_dy1[idx] + d_dy2[idx]) / 2.0f;


	float u_avg = uv[idx].x, v_avg = uv[idx].y, num = 0, denom = 0.01 + grad_x * grad_x + grad_y * grad_y;
	int count = 0;
	for (int i = -1; i <= 1; i++) {
		for (int j = -1; j <= 1; j++) {
			u_avg += uv[GIDX(row + i, col + j, H, W)].x * hs_kernel[count];
			v_avg += uv[GIDX(row + i, col + j, H, W)].y * hs_kernel[count];
			count++;
		}
	}
	num = (grad_x * u_avg + grad_y * v_avg + d_dt[idx]);

	uv[idx].x = u_avg - (grad_x * num) / denom;
	uv[idx].y = v_avg - (grad_y * num) / denom;

	uv1[idx].x = uv[idx].x;
	uv1[idx].y = uv[idx].y;
}

void launch_optflow(float* d_dx1, float* d_dy1, float* d_dx2, float* d_dy2, float* d_dt, float4* uv, float4* uv1, int H, int W) {

	dim3 blockSize = { 32, 32 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(W / blockSize.x)),
		static_cast<unsigned int>(ceil(H / blockSize.y))
	};

	const float k[9] = { 0.084, 0.167, 0.084, 0.167, -1, 0.167, 0.084, 0.167, 0.084 };

	setHS(k, 9);

	kernel_optflow << <gridSize, blockSize >> > (
		d_dx1, d_dy1, d_dx2, d_dy2, d_dt, uv, uv1, H, W
		);

	CUDACHECK(hipDeviceSynchronize());
};

__global__ void kernel_fill(float4* d_dx1, float val, int numel) {
	size_t col = threadIdx.x + blockIdx.x * blockDim.x;
	if (col >= numel) { return; }

	d_dx1[col] = make_float4(val, val, val, val);
}

void launch_fill(float4* d_dx1, float val, int numel) {
	dim3 blockSize = { 320 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(numel / blockSize.x))
	};
	kernel_fill << <gridSize, blockSize >> > (
		d_dx1, val, numel
		);

	CUDACHECK(hipDeviceSynchronize());
};


__global__ void kernel_blur(float4* d_I, float4* d_Ib, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = GIDX(row, col, H, W);

	if (row >= H - KERN_RADIUS || row <= KERN_RADIUS || col >= W - KERN_RADIUS || col <= KERN_RADIUS) {
		return;
	}


	d_Ib[idx] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	int count = 0;
	for (int i = -KERN_RADIUS; i <= KERN_RADIUS; i++) {
		for (int j = -KERN_RADIUS; j <= KERN_RADIUS; j++) {
			d_Ib[idx].x += d_I[GIDX(row + i, col + j, H, W)].x * gaussian_kernel[count];
			d_Ib[idx].y += d_I[GIDX(row + i, col + j, H, W)].y * gaussian_kernel[count];
			d_Ib[idx].z += d_I[GIDX(row + i, col + j, H, W)].z * gaussian_kernel[count];
			count++;
		}
	}

}




void launch_blur(float4* d_I, float4* d_Ib, int H, int W) {

	dim3 blockSize = { 32, 32 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(W / blockSize.x)),
		static_cast<unsigned int>(ceil(H / blockSize.y))
	};

	float a = 1 / 256.0;
	float b = 4 / 256.0;
	float c = 6 / 256.0;
	float d = 16 / 256.0;
	float e = 24 / 256.0;
	float f = 36 / 256.0;

	const float k[25] = {
		a, b, c, b, a,
		b, d, e, d, b,
		c, e, f, e, c,
		b, d, e, d, b,
		a, b, c, b, a
	};

	setGaussian(k, 25);

	kernel_blur << <gridSize, blockSize >> > (
		d_I, d_Ib, H, W
		);

	CUDACHECK(hipDeviceSynchronize());
};


__global__ void kernel_convection(float4* d_uv1, float4* d_uv2, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = GIDX(row, col, H, W);

	if (row >= H - KERN_RADIUS || row <= KERN_RADIUS || col >= W - KERN_RADIUS || col <= KERN_RADIUS) {
		return;
	}

	d_uv2[idx].x = d_uv1[idx].x -
		(1 / 30.0f * (d_uv1[idx].x * (d_uv1[idx].x - d_uv1[GIDX(row - 1, col, H, W)].x))) -
		(1 / 30.0f * (d_uv1[idx].y * (d_uv1[idx].x - d_uv1[GIDX(row, col - 1, H, W)].x))) +
		(0.05 * (1 / 30.0f) * (d_uv1[GIDX(row + 1, col, H, W)].x - 2 * d_uv1[idx].x + d_uv1[GIDX(row - 1, col, H, W)].x)) +
		(0.05 * (1 / 30.0f) * (d_uv1[GIDX(row, col + 1, H, W)].x - 2 * d_uv1[idx].x + d_uv1[GIDX(row, col - 1, H, W)].x));

	d_uv2[idx].y = d_uv1[idx].y -
		(1 / 30.0f * (d_uv1[idx].x * (d_uv1[idx].y - d_uv1[GIDX(row - 1, col, H, W)].y))) -
		(1 / 30.0f * (d_uv1[idx].y * (d_uv1[idx].y - d_uv1[GIDX(row, col - 1, H, W)].y))) +
		(0.05 * (1 / 30.0f) * (d_uv1[GIDX(row + 1, col, H, W)].y - 2 * d_uv1[idx].y + d_uv1[GIDX(row - 1, col, H, W)].y)) +
		(0.05 * (1 / 30.0f) * (d_uv1[GIDX(row, col + 1, H, W)].y - 2 * d_uv1[idx].y + d_uv1[GIDX(row, col - 1, H, W)].y));



	d_uv1[idx].x = d_uv2[idx].x;
	d_uv1[idx].y = d_uv2[idx].y;

}





void launch_convection(float4* d_uv1, float4* d_uv2, int H, int W) {

	dim3 blockSize = { 32, 32 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(W / blockSize.x)),
		static_cast<unsigned int>(ceil(H / blockSize.y))
	};

	kernel_convection << <gridSize, blockSize >> > (
		d_uv1, d_uv2, H, W
		);

	CUDACHECK(hipDeviceSynchronize());

};

__global__ void kernel_convert(uchar3* d_Iin, float4* d_Iout, int numel) {
	size_t col = threadIdx.x + blockIdx.x * blockDim.x;
	if (col >= numel) { return; }
	uchar3 val = d_Iin[col];

	d_Iout[col] = make_float4(
		val.x / 255.0f,
		val.y / 255.0f,
		val.z / 255.0f,
		1.0f
	);
}

void launch_convert(uchar3* d_Iin, float4* d_Iout, int numel) {
	dim3 blockSize = { 320 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(numel / blockSize.x))
	};
	kernel_convert << <gridSize, blockSize >> > (
		d_Iin, d_Iout, numel
		);

	CUDACHECK(hipDeviceSynchronize());
};
