#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
__device__ inline size_t rcToidx(size_t row, size_t col, size_t H, size_t W) {

	return row * W + col;
}

__global__ void kernel_partials(
	uchar3* d_f1ptr,
	float* d_f1dx, float* d_f1dy,
	int H, int W
) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = rcToidx(row, col, H, W);
	if (row >= H - 5 || row <= 5 || col >= W-1 || col <= 5) {
		return;
	}

	//printf("%f %f %f", d_f1ptr[idx].x, d_f1dx[idx], d_f1dy[idx]);
	float gray_x1 = 0.2989f * (d_f1ptr[rcToidx(row, col - 1, H, W)].x/255.0f) + 
					0.5870f * (d_f1ptr[rcToidx(row, col - 1, H, W)].y/255.0f) + 
					0.1140f * (d_f1ptr[rcToidx(row, col - 1, H, W)].z/255.0f);
	//printf("%d %d %f \n", row, col, gray_x1);
	float gray_x2 = 0.2989f * (d_f1ptr[rcToidx(row, col + 1, H, W)].x/255.0f) + 
					0.5870f * (d_f1ptr[rcToidx(row, col + 1, H, W)].y/255.0f) + 
					0.1140f * (d_f1ptr[rcToidx(row, col + 1, H, W)].z/255.0f);

	float gray_y1 = 0.2989f * (d_f1ptr[rcToidx(row - 1, col, H, W)].x/255.0f) + 
					0.5870f * (d_f1ptr[rcToidx(row - 1, col, H, W)].y/255.0f) + 
					0.1140f * (d_f1ptr[rcToidx(row - 1, col, H, W)].z/255.0f);

	float gray_y2 = 0.2989f * (d_f1ptr[rcToidx(row + 1, col, H, W)].x/255.0f) + 
					0.5870f * (d_f1ptr[rcToidx(row + 1, col, H, W)].y/255.0f) + 
					0.1140f * (d_f1ptr[rcToidx(row + 1, col, H, W)].z/255.0f);

	
	
	d_f1dx[idx] = (gray_x1 - gray_x2)/2.0f;
	d_f1dy[idx] = (gray_y1 - gray_y2)/2.0f;
	

}


void launch_partials(
	uchar3* d_f1ptr, 
	float* d_f1dx, float* d_f1dy, 
	int H, int W
) {

	const dim3 blockSize(32, 32);
	const dim3 gridSize(H / blockSize.x + 1, W / blockSize.y + 1);

	kernel_partials << <gridSize, blockSize >> > (
		d_f1ptr, d_f1dx, d_f1dy, H, W
	);

	hipError_t error = hipDeviceSynchronize();
	std::cout << error << "\n";
}


__global__ void kernel_sub(uchar3* d_f1ptr, uchar3* d_f2ptr, float* d_dt, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = rcToidx(row, col, H, W);

	if (row >= H - 5 || row <= 5 || col >= W - 5 || col <= 5) {
		return;
	}

	float gray_1 =	0.2989f * (d_f1ptr[idx].x / 255.0f) +
					0.5870f * (d_f1ptr[idx].y / 255.0f) +
					0.1140f * (d_f1ptr[idx].z / 255.0f);

	float gray_2 =	0.2989f * (d_f2ptr[idx].x / 255.0f) +
					0.5870f * (d_f2ptr[idx].y / 255.0f) +
					0.1140f * (d_f2ptr[idx].z / 255.0f);


	d_dt[idx] = gray_2 - gray_1;

}


void launch_sub(uchar3* d_f1ptr, uchar3* d_f2ptr, float* d_dt, int H, int W) {

	const dim3 blockSize(32, 32);
	const dim3 gridSize(H / blockSize.x + 1, W / blockSize.y + 1);

	kernel_sub << <gridSize, blockSize >> > (
		d_f1ptr, d_f2ptr, d_dt, H, W
	);

	hipError_t error = hipDeviceSynchronize();
	std::cout << error << "\n";
}

__global__ void kernel_optflow(float* d_dx1, float* d_dy1, float* d_dx2, float* d_dy2, float* d_dt, float4* uv, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = rcToidx(row, col, H, W);
	if (row >= H - 5 || row <= 5 || col >= W-5 || col <= 5) {
		return;
	}

	float grad_x = d_dx1[idx] + d_dx2[idx];
	float grad_y = d_dy1[idx] + d_dy2[idx];

	float magnitude = sqrtf(grad_x*grad_x + grad_y * grad_y + 0.1);
	uv[idx].x = d_dt[idx] * (grad_x / magnitude);
	uv[idx].y = d_dt[idx] * (grad_y / magnitude);

}

void launch_optflow(float* d_dx1, float* d_dy1, float* d_dx2, float* d_dy2, float* d_dt, float4* uv, int H, int W) {

	const dim3 blockSize(32, 32);
	const dim3 gridSize(H / blockSize.x + 1, W / blockSize.y + 1);

	kernel_optflow << <gridSize, blockSize >> > (
		d_dx1, d_dy1, d_dx2, d_dy2, d_dt, uv, H, W
	);

	hipError_t error = hipDeviceSynchronize();
	std::cout << error << "\n";
};