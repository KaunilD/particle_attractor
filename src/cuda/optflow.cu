#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
__device__ inline size_t rcToidx(size_t row, size_t col, size_t H, size_t W) {

	return row * W + col;
}

__global__ void kernel_partials(
	uchar3* d_f1ptr,
	float* d_f1dx, float* d_f1dy,
	int H, int W
) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = rcToidx(row, col, H, W);
	if (row >= H - 5 || row <= 5 || col >= W - 1 || col <= 5) {
		return;
	}

	//printf("%f %f %f", d_f1ptr[idx].x, d_f1dx[idx], d_f1dy[idx]);
	float gray_x1 = 0.2989f * (d_f1ptr[rcToidx(row, col - 1, H, W)].x / 255.0f) +
		0.5870f * (d_f1ptr[rcToidx(row, col - 1, H, W)].y / 255.0f) +
		0.1140f * (d_f1ptr[rcToidx(row, col - 1, H, W)].z / 255.0f);
	//printf("%d %d %f \n", row, col, gray_x1);
	float gray_x2 = 0.2989f * (d_f1ptr[rcToidx(row, col + 1, H, W)].x / 255.0f) +
		0.5870f * (d_f1ptr[rcToidx(row, col + 1, H, W)].y / 255.0f) +
		0.1140f * (d_f1ptr[rcToidx(row, col + 1, H, W)].z / 255.0f);

	float gray_y1 = 0.2989f * (d_f1ptr[rcToidx(row - 1, col, H, W)].x / 255.0f) +
		0.5870f * (d_f1ptr[rcToidx(row - 1, col, H, W)].y / 255.0f) +
		0.1140f * (d_f1ptr[rcToidx(row - 1, col, H, W)].z / 255.0f);

	float gray_y2 = 0.2989f * (d_f1ptr[rcToidx(row + 1, col, H, W)].x / 255.0f) +
		0.5870f * (d_f1ptr[rcToidx(row + 1, col, H, W)].y / 255.0f) +
		0.1140f * (d_f1ptr[rcToidx(row + 1, col, H, W)].z / 255.0f);



	d_f1dx[idx] = (gray_x1 - gray_x2) / 2.0f;
	d_f1dy[idx] = (gray_y1 - gray_y2) / 2.0f;


}


void launch_partials(
	uchar3* d_f1ptr,
	float* d_f1dx, float* d_f1dy,
	int H, int W
) {

	const dim3 blockSize(32, 32);
	const dim3 gridSize(H / blockSize.x + 1, W / blockSize.y + 1);

	kernel_partials << <gridSize, blockSize >> > (
		d_f1ptr, d_f1dx, d_f1dy, H, W
		);

	hipError_t error = hipDeviceSynchronize();
	std::cout << error << "\n";
}


__global__ void kernel_sub(uchar3* d_f1ptr, uchar3* d_f2ptr, float* d_dt, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = rcToidx(row, col, H, W);

	if (row >= H - 5 || row <= 5 || col >= W - 5 || col <= 5) {
		return;
	}

	float gray_1 = 0.2989f * (d_f1ptr[idx].x / 255.0f) +
		0.5870f * (d_f1ptr[idx].y / 255.0f) +
		0.1140f * (d_f1ptr[idx].z / 255.0f);

	float gray_2 = 0.2989f * (d_f2ptr[idx].x / 255.0f) +
		0.5870f * (d_f2ptr[idx].y / 255.0f) +
		0.1140f * (d_f2ptr[idx].z / 255.0f);


	d_dt[idx] = gray_2 - gray_1;

}


void launch_sub(uchar3* d_f1ptr, uchar3* d_f2ptr, float* d_dt, int H, int W) {

	const dim3 blockSize(32, 32);
	const dim3 gridSize(H / blockSize.x + 1, W / blockSize.y + 1);

	kernel_sub << <gridSize, blockSize >> > (
		d_f1ptr, d_f2ptr, d_dt, H, W
		);

	hipError_t error = hipDeviceSynchronize();
	std::cout << error << "\n";
}

__global__ void kernel_optflow(float* d_dx1, float* d_dy1, float* d_dx2, float* d_dy2, float* d_dt, float4* uv, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = rcToidx(row, col, H, W);
	if (row >= H - 5 || row <= 5 || col >= W - 5 || col <= 5) {
		return;
	}

	float grad_x = d_dx1[idx] + d_dx2[idx];
	float grad_y = d_dy1[idx] + d_dy2[idx];

	float k[9] = { 0.084, 0.167, 0.084, 0.167, -1, 0.167, 0.084, 0.167, 0.084 };

	float u_avg = uv[idx].x, v_avg = uv[idx].y, num = 0, denom = 0.0001 + grad_x * grad_x + grad_y * grad_y;
	int count = 0;
	for (int i = -1; i <= 1; i++) {
		for (int j = -1; j <= 1; j++) {
			u_avg += uv[rcToidx(row + i, col + j, H, W)].x * k[count];
			v_avg += uv[rcToidx(row + i, col + j, H, W)].y * k[count];
			count++;
		}
	}
	num = (grad_x * u_avg + grad_y * v_avg + d_dt[idx]);
	uv[idx].x = u_avg - (grad_x * num) / denom;
	uv[idx].y = v_avg - (grad_y * num) / denom;
}

void launch_optflow(float* d_dx1, float* d_dy1, float* d_dx2, float* d_dy2, float* d_dt, float4* uv, int H, int W) {

	const dim3 blockSize(32, 32);
	const dim3 gridSize(H / blockSize.x + 1, W / blockSize.y + 1);

	kernel_optflow << <gridSize, blockSize >> > (
		d_dx1, d_dy1, d_dx2, d_dy2, d_dt, uv, H, W
		);

	hipError_t error = hipDeviceSynchronize();
	std::cout << error << "\n";
};

__global__ void kernel_fill(float4* d_dx1, float val, int numel) {
	size_t col = threadIdx.x + blockIdx.x * blockDim.x;
	if (col >= numel) { return; }
	
	d_dx1[col] = make_float4(val, val, val, val);
}

void launch_fill(float4* d_dx1, float val, int numel) {

	const dim3 blockSize(1024);
	const dim3 gridSize(numel / blockSize.x , 1, 1);

	kernel_fill << <gridSize, blockSize >> > (
		d_dx1, val, numel
		);

	hipError_t error = hipDeviceSynchronize();
	std::cout << error << "\n";
};