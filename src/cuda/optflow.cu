#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cuda/utils.hpp"

#define KERN_RADIUS 5

__constant__ float gaussian_kernel[KERN_RADIUS * KERN_RADIUS];
__constant__ float hs_kernel[9];

__device__ size_t GIDX(size_t row, size_t col, int H, int W) {
	return row * W + col;
}

void setGaussian(float const* elements, const int count) {
	if (count != 25) {
		std::cerr << "Only 5x5 gaussian kernel supported!" << std::endl;
		return;
	}
	CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(gaussian_kernel), elements, count * sizeof(float)));
}

void setHS(float const* elements, const int count) {
	if (count != 9) {
		std::cerr << "Only 3x3 horn-schunke kernel supported!" << std::endl;
		return;
	}
	CUDACHECK(hipMemcpyToSymbol(HIP_SYMBOL(hs_kernel), elements, count * sizeof(float)));
}

__global__ void kernel_grey(
	float4* d_Iin,
	float* d_Iout,
	int numel
) {

	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	if (col >= numel) {
		return;
	}

	float4 pixel = d_Iin[col];

	d_Iout[col] = 0.2989f * (pixel.x) + 0.5870f * (pixel.y) + 0.1140f * (pixel.z);
}

void launch_gray(
	float4* d_Iin,
	float* d_Iout,
	int numel
) {


	dim3 blockSize = { 320 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(numel / blockSize.x))
	};

	kernel_grey << <gridSize, blockSize >> > (
		d_Iin, d_Iout, numel
		);

	CUDACHECK(hipDeviceSynchronize());
}


__global__ void kernel_partials(
	float* d_f1ptr,
	float* d_f1dx, float* d_f1dy,
	int H, int W
) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;

	size_t idx = GIDX(row, col, H, W);
	if (row >= H || row <= 1 || col >= W || col <= 1) {
		return;
	}

	float gray_x1 = d_f1ptr[GIDX(row, col - 1, H, W)];
	float gray_x2 = d_f1ptr[GIDX(row, col + 1, H, W)];

	float gray_y1 = d_f1ptr[GIDX(row - 1, col, H, W)];
	float gray_y2 = d_f1ptr[GIDX(row + 1, col, H, W)];

	d_f1dx[idx] = (gray_x2 - gray_x1) / 2.0f;
	d_f1dy[idx] = (gray_y2 - gray_y1) / 2.0f;
}


void launch_partials(
	float* d_f1ptr,
	float* d_f1dx, float* d_f1dy,
	int H, int W
) {


	dim3 blockSize = { 32, 32 };
	dim3 gridSize = {
	static_cast<unsigned int>(ceil(W / blockSize.x)),
	static_cast<unsigned int>(ceil(H / blockSize.y))
	};

	kernel_partials << <gridSize, blockSize >> > (
		d_f1ptr, d_f1dx, d_f1dy, H, W
		);

	CUDACHECK(hipDeviceSynchronize());
}


__global__ void kernel_sub(float* d_f1ptr, float* d_f2ptr, float* d_dt, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = GIDX(row, col, H, W);

	if (row >= H || col >= W) {
		return;
	}

	d_dt[idx] = d_f2ptr[idx] - d_f1ptr[idx];

}


void launch_sub(float* d_f1ptr, float* d_f2ptr, float* d_dt, int H, int W) {

	dim3 blockSize = { 32, 32 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(W / blockSize.x)),
		static_cast<unsigned int>(ceil(H / blockSize.y))
	};

	kernel_sub << <gridSize, blockSize >> > (
		d_f1ptr, d_f2ptr, d_dt, H, W
		);

	CUDACHECK(hipDeviceSynchronize());
}

__global__ void kernel_optflow(float* d_dx1, float* d_dy1, float* d_dx2, float* d_dy2, float* d_dt, float4* uv, float4* uv1, int H, int W) {

	const size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	const size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	const size_t idx = GIDX(row, col, H, W);


	if (row >= H - 2 || row <= 2 || col >= W - 2 || col <= 2) {
		return;
	}
	__syncthreads();
	

	float dx2 = 0.0f, dy2 = 0.0f;
	float dxdy = 0.0f;
	float dxdt = 0.0f, dydt = 0.0f;

	for (int i = -2; i <= 2; i++) {
		for (int j = -2; j <= 2; j++) {
			dx2 += d_dx1[GIDX(row + i, col + j, H, W)] * d_dx1[GIDX(row + i, col + j, H, W)];
			dy2 += d_dy1[GIDX(row + i, col + j, H, W)] * d_dy1[GIDX(row + i, col + j, H, W)];
			
			dxdy += d_dx1[GIDX(row + i, col + j, H, W)] * d_dy1[GIDX(row + i, col + j, H, W)];
			
			dxdt += d_dx1[GIDX(row + i, col + j, H, W)] * d_dt[GIDX(row + i, col + j, H, W)];
			dydt += d_dy1[GIDX(row + i, col + j, H, W)] * d_dt[GIDX(row + i, col + j, H, W)];
		}
	}

	__syncthreads();
	float det = dx2 * dy2 - (dxdy * dxdy);
	if (abs(det) <= 1.5e-8) { // 1.5e-5 is based on 1/(255*255)
		uv[idx].x = 0.0f;
		uv[idx].y = 0.0f;
		uv1[idx] = uv[idx];
		return;
	}

	__syncthreads();
	float trace = dx2 + dy2; 
	float delta = sqrtf(trace * trace - 4.0f * det); // delta x2

	if (isnan(delta) || trace - delta <= 0.0002) {
		uv[idx].x = 0.0f;
		uv[idx].y = 0.0f;
		uv1[idx] = uv[idx];
		return;
	}

	__syncthreads();
	// Calculate flow components
	
	uv[idx].x = (dy2 * -dxdt + dxdy * dydt)/det; 
	uv[idx].y = (dxdy * dxdt - dx2 * dydt)/ det;
	uv1[idx] = uv[idx];

}

void launch_optflow(float* d_dx1, float* d_dy1, float* d_dx2, float* d_dy2, float* d_dt, float4* uv, float4* uv1, int H, int W) {

	dim3 blockSize = { 16, 16 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(W / blockSize.x)),
		static_cast<unsigned int>(ceil(H / blockSize.y))
	};
	size_t sharedMem = sizeof(float4) * (blockSize.y + 2) * (blockSize.x + 2);

	const float k[9] = { 0.084, 0.167, 0.084, 0.167, -1, 0.167, 0.084, 0.167, 0.084 };

	setHS(k, 9);

	kernel_optflow << <gridSize, blockSize, sharedMem >> > (
		d_dx1, d_dy1, d_dx2, d_dy2, d_dt, uv, uv1, H, W
		);

	CUDACHECK(hipDeviceSynchronize());
};

__global__ void kernel_fill(float4* d_dx1, float val, int numel) {
	size_t col = threadIdx.x + blockIdx.x * blockDim.x;
	if (col >= numel) { return; }

	d_dx1[col].x = val;
	d_dx1[col].y = val;
	d_dx1[col].z = val;
	d_dx1[col].w = val;
}

void launch_fill(float4* d_dx1, float val, int numel) {
	dim3 blockSize = { 320 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(numel / blockSize.x))
	};
	kernel_fill << <gridSize, blockSize >> > (
		d_dx1, val, numel
		);

	CUDACHECK(hipDeviceSynchronize());
};


__global__ void kernel_blur(float* d_I, float* d_Ib, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = GIDX(row, col, H, W);

	if (row >= H - KERN_RADIUS || row <= KERN_RADIUS || col >= W - KERN_RADIUS || col <= KERN_RADIUS) {
		return;
	}

	int count = 0;
	for (int i = -KERN_RADIUS; i <= KERN_RADIUS; i++) {
		for (int j = -KERN_RADIUS; j <= KERN_RADIUS; j++) {
			d_Ib[idx] += d_I[GIDX(row + i, col + j, H, W)] * gaussian_kernel[count];
			count++;
		}
	}

}

void launch_blur(float* d_I, float* d_Ib, int H, int W) {

	dim3 blockSize = { 32, 32 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(W / blockSize.x)),
		static_cast<unsigned int>(ceil(H / blockSize.y))
	};

	float a = 1 / 256.0;
	float b = 4 / 256.0;
	float c = 6 / 256.0;
	float d = 16 / 256.0;
	float e = 24 / 256.0;
	float f = 36 / 256.0;

	const float k[25] = {
		a, b, c, b, a,
		b, d, e, d, b,
		c, e, f, e, c,
		b, d, e, d, b,
		a, b, c, b, a
	};

	setGaussian(k, 25);

	kernel_blur << <gridSize, blockSize >> > (
		d_I, d_Ib, H, W
		);

	CUDACHECK(hipDeviceSynchronize());
};


__global__ void kernel_convection(float4* d_uv1, float4* d_uv2, float* d_p, int H, int W) {

	size_t row = threadIdx.y + blockDim.y * blockIdx.y;
	size_t col = threadIdx.x + blockDim.x * blockIdx.x;
	size_t idx = GIDX(row, col, H, W);


	if (row >= H - 1 || row <= 1 || col >= W - 1 || col <= 1) {
		return;
	}

	float dt = 0.0001;
	float nu = 0.1;
	float dx = 1, dy = 1, dx2 = dx*dx, dy2 = dy*dy;
	float rho = 1.2;
	
	float4 d_uv1r1 = d_uv1[GIDX(row - 1, col, H, W)], d_uv1r2 = d_uv1[GIDX(row + 1, col, H, W)];
	float4 d_uv1c1 = d_uv1[GIDX(row, col - 1, H, W)], d_uv1c2 = d_uv1[GIDX(row, col + 1, H, W)];
	float4 d_uv1_idx = d_uv1[idx];

	float d_pr1 = d_p[GIDX(row - 1, col, H, W)], d_pr2 = d_p[GIDX(row + 1, col, H, W)];
	float d_pc1 = d_p[GIDX(row, col - 1, H, W)], d_pc2 = d_p[GIDX(row, col + 1, H, W)];
	float d_p_idx = d_p[idx];
	float a = (d_pr2 + d_pr1) * dy2;
	float b = (d_pc2 + d_pc1) * dx2;
	float c = 2 * (dx2 + dy2);
	float d = rho * dx2 * dy2;

	d_p[idx] = ((a + b) / c) - (d / c) *
		(1 / dt * ((d_uv1r2.x - d_uv1r1.x) / (2 * dx) + (d_uv1c2.y - d_uv1c1.y) / (2 * dy)) -
			((d_uv1r2.x - d_uv1r1.x) / (2 * dx)) * ((d_uv1c2.y - d_uv1c1.y) / (2 * dy)) -
			(2 * (d_uv1c2.x - d_uv1c1.x) / (2 * dy)) * (2 * (d_uv1r2.y - d_uv1r1.y) / (2 * dx)) -
			(2 * (d_uv1c2.y - d_uv1c1.y) / (2 * dy)) * (2 * (d_uv1c2.y - d_uv1c1.y) / (2 * dx)));


	d_uv2[idx].x = d_uv1_idx.x -
		d_uv1_idx.x * dt * (d_uv1_idx.x - d_uv1r1.x) -
		d_uv1_idx.y * dt * (d_uv1_idx.x - d_uv1c1.x) -
		(dt / rho) * (d_pr2 - d_pr1) + 
		nu * (dt * (d_uv1r2.x - 2 * d_uv1[idx].x + d_uv1r1.x) + dt * (d_uv1c2.x - 2 * d_uv1_idx.x + d_uv1c1.x));

	d_uv2[idx].y = d_uv1_idx.y -
		d_uv1_idx.x * dt * (d_uv1_idx.y - d_uv1r1.y) -
		d_uv1_idx.y * dt * (d_uv1_idx.y - d_uv1c1.y) -
		(dt / rho) * (d_pr2 - d_pr1) +
		nu * (dt * (d_uv1r2.y - 2 * d_uv1_idx.y + d_uv1r1.y) +
			dt * (d_uv1c2.y - 2 * d_uv1_idx.y + d_uv1c1.y));
}





void launch_convection(float4* d_uv1, float4* d_uv2, float*p, int H, int W) {

	dim3 blockSize = { 32, 32 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(W / blockSize.x)),
		static_cast<unsigned int>(ceil(H / blockSize.y))
	};
	int padding = 1;

	size_t s_numel = (blockSize.x + 2 * padding) * (blockSize.y + 2 * padding);
	size_t s_uv1 = s_numel * sizeof(float4);
	size_t s_p = s_numel * sizeof(float);

	kernel_convection << <gridSize, blockSize, s_uv1 + s_p >> > (
		d_uv1, d_uv2, p, H, W
		);

	CUDACHECK(hipDeviceSynchronize());

};

__global__ void kernel_convert(uchar3* d_Iin, float4* d_Iout, int numel) {
	size_t col = threadIdx.x + blockIdx.x * blockDim.x;
	if (col >= numel) { return; }
	uchar3 val = d_Iin[col];

	d_Iout[col] = make_float4(
		val.x / 255.0f,
		val.y / 255.0f,
		val.z / 255.0f,
		1.0f
	);
}

void launch_convert(uchar3* d_Iin, float4* d_Iout, int numel) {
	dim3 blockSize = { 320 };
	dim3 gridSize = {
		static_cast<unsigned int>(ceil(numel / blockSize.x))
	};
	kernel_convert << <gridSize, blockSize >> > (
		d_Iin, d_Iout, numel
		);

	CUDACHECK(hipDeviceSynchronize());
};
